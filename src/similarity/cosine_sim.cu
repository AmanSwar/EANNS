#include "hip/hip_runtime.h"
#include <cstddef>
#include <hipblas.h>

#include "../../include/similarity/cosine_sim.hpp"

/*
Cosine Similarity -> 
vector A , vector B
nominator -> A * B
denominator -> norm(A) * norm(B)

*/


float cosine_similarity(float* h_vectorA , float* h_vectorB , int n_dim){
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float *d_vectorA , *d_vectorB;

    size_t size = n_dim * sizeof(float);

    hipMalloc(&d_vectorA , size);
    hipMalloc(&d_vectorB , size);

    hipMemcpy(d_vectorA , h_vectorA , size , hipMemcpyHostToDevice);
    hipMemcpy(d_vectorB , h_vectorB , size , hipMemcpyHostToDevice);
    
    float norm_a , norm_b;
    hipblasSnrm2(handle , n_dim , d_vectorA ,1 , &norm_a);
    hipblasSnrm2(handle , n_dim , d_vectorB ,1 , &norm_b);

    float dot_product_value;
    hipblasSdot(handle , n_dim , d_vectorA , 1 , d_vectorB ,1 , &dot_product_value);

    float similarity = dot_product_value / (norm_a * norm_b);

    return similarity;

}